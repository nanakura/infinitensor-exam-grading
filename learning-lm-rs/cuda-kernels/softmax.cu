#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

template<typename T>
__device__ T maximum(T a, T b) {
    return (a > b) ? a : b;
}

template<typename T>
__device__ T zero() {
    return T(0.0f);
}

template<typename T>
__device__ T exp_wrapper(T x) {
    return expf(x);
}

template<>
__device__ __half exp_wrapper(__half x) {
    return hexp(x);
}

template<>
__device__ __hip_bfloat16 exp_wrapper(__hip_bfloat16 x) {
    return hexp(x);
}

extern "C"__global__ void masked_softmax_f32(
    float* data,
    const int batch,
    const int seq_len,
    const int total_seq_len
) {
    int b = blockIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (b < batch && i < seq_len) {
        int base = b * seq_len * total_seq_len;
        int offset = base + i * total_seq_len;
        int boundary = total_seq_len - seq_len + i + 1;
        
        // Find max
        float max_val = data[offset];
        for (int j = 1; j < boundary; j++) {
            max_val = maximum(max_val, data[offset + j]);
        }
        
        // Compute exp and sum
        float sum = zero<float>();
        for (int j = 0; j < boundary; j++) {
            float val = exp_wrapper(data[offset + j] - max_val);
            data[offset + j] = val;
            sum += val;
        }
        
        // Normalize and mask
        for (int j = 0; j < boundary; j++) {
            data[offset + j] = data[offset + j] / sum;
        }
        for (int j = boundary; j < total_seq_len; j++) {
            data[offset + j] = zero<float>();
        }
    }
}