#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_fp16.h"
#include "hip/hip_bf16.h"
#include<stdint.h>

template<typename T>
__device__ T maximum(T a, T b) {
    return (a > b) ? a : b;
}

template<typename T>
__device__ T exp_wrapper(T x) {
    return expf(x);
}

template<>
__device__ __half exp_wrapper(__half x) {
    return hexp(x);
}

template<>
__device__ __hip_bfloat16 exp_wrapper(__hip_bfloat16 x) {
    return hexp(x);
}

template <typename T>
__device__ void random_sample(
    T* logits,           
    unsigned int* result,   
    float top_p,            
    unsigned int top_k,
    float temperature,
    unsigned int seed,
    int size
) {
    __shared__ T max_val;
    __shared__ T sum_exp;
    __shared__ T probs[4096];
    
    T local_max = -INFINITY;
    for(int i = threadIdx.x; i < size; i += blockDim.x) {
        local_max = maximum(local_max, logits[i]);
    }
    
    __shared__ T temp_max[256];
    temp_max[threadIdx.x] = local_max;
    __syncthreads();
    
    for(int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if(threadIdx.x < stride) {
            temp_max[threadIdx.x] = maximum(temp_max[threadIdx.x], temp_max[threadIdx.x + stride]);
        }
        __syncthreads();
    }
    
    if(threadIdx.x == 0) {
        max_val = temp_max[0];
        sum_exp = T(0.0f);
    }
    __syncthreads();

    for(int i = threadIdx.x; i < size; i += blockDim.x) {
        T val = exp_wrapper((logits[i] - max_val) / T(temperature));
        probs[i] = val;
        atomicAdd(&sum_exp, val);
    }
    __syncthreads();
    
    for(int i = threadIdx.x; i < size; i += blockDim.x) {
        probs[i] /= sum_exp;
    }
    __syncthreads();
    
    if(threadIdx.x == 0) {
        int indices[4096];
        T sorted_probs[4096];
        for(int i = 0; i < size; i++) {
            indices[i] = i;
            sorted_probs[i] = probs[i];
        }
        
        int k = min((int)top_k, size);
        for(int i = 0; i < k; i++) {
            for(int j = i + 1; j < size; j++) {
                if(sorted_probs[j] > sorted_probs[i]) {
                    float temp_prob = sorted_probs[i];
                    sorted_probs[i] = sorted_probs[j];
                    sorted_probs[j] = temp_prob;
                    int temp_idx = indices[i];
                    indices[i] = indices[j];
                    indices[j] = temp_idx;
                }
            }
        }
        
        T cumsum = T(0.0f);
        int last_idx = k - 1;
        if(top_p < 1.0f) {
            for(int i = 0; i < k; i++) {
                cumsum += sorted_probs[i];
                if(cumsum >= T(top_p)) {
                    last_idx = i;
                    break;
                }
            }
        }
        
        cumsum = T(0.0f);
        for(int i = 0; i <= last_idx; i++) {
            cumsum += sorted_probs[i];
        }
        for(int i = 0; i <= last_idx; i++) {
            sorted_probs[i] /= cumsum;
        }
        
        hiprandState state;
        hiprand_init(seed, 0, 0, &state);
        T rand_val = T(hiprand_uniform(&state));
        
        cumsum = T(0.0f);
        for(int i = 0; i <= last_idx; i++) {
            cumsum += sorted_probs[i];
            if(rand_val <= cumsum) {
                *result = indices[i];
                break;
            }
        }
        if(rand_val > cumsum) {
            *result = indices[last_idx];
        }
    }
}

#define OPS(TYPENAME, RUST_NAME) \
    extern "C" __global__ void random_sample_##RUST_NAME( \
        TYPENAME* logits, \
        unsigned int* result, \
        float top_p, \
        unsigned int top_k, \
        float temperature, \
        unsigned int seed, \
        int size \
    ) { \
        random_sample<TYPENAME>(logits, result, top_p, top_k, temperature, seed, size); \
    } \

#if __CUDA_ARCH__ >= 800
OPS(__nv_bfloat16, bf16)
#endif

#if __CUDA_ARCH__ >= 530
OPS(__half, f16)
#endif

OPS(float, f32)
OPS(double, f64)